#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//
// Smurf
// =====
// ##### Martin Kirst, Johannes Jendersie, Christoph L�mmerhirt, Laura Osten #####
//
// Smoke Surfaces: An Interactive Flow Visualization
// Technique Inspired by Real-World Flow Experiments
//
// File:              /src/cudaintegration.cu
// Author:            Christoph L�mmerhirt
// Creation Date:     2012.01.11
// Description:
//
// Declaration of the interface from C++ to Cuda and the Cuda-Kernel.
//
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// Preprocessor Directives and Namespaces
////////////////////////////////////////////////////////////////////////////////

#include <math.h>
#include "cudamath.hpp"

__device__ int3 convert_int3(float3 vec)
{
	int3 tmp;
	tmp.x=(int)vec.x;
	tmp.y=(int)vec.y;
	tmp.z=(int)vec.z;
	return tmp;
}

__device__ float3 convert_float3(int3 vec)
{
	float3 tmp;
	tmp.x=(int)vec.x;
	tmp.y=(int)vec.y;
	tmp.z=(int)vec.z;
	return tmp;
}

__device__ float3 Sample(float3 Vector, const float *Vector_Field, int3 Size)
{
	int3 fi;
	int index;

	fi=convert_int3(Vector);
	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
			return make_float3(0,0,0);

	index=fi.x+fi.y*Size.x+fi.z*Size.y*Size.x;

	return make_float3(Vector_Field[index+0],Vector_Field[index+1],Vector_Field[index+2]);
}

__device__ float3 lerp(float3 start, float3 end, float t)
{
	return start+t*(end-start);
}

__device__ float3 SampleL(float3 Vector, const float *Vector_Field, int3 Size)
{
	float3 s[8];

	int3 fi;
	fi=convert_int3(Vector);

	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
			return make_float3(0,0,0);

	Vector-=convert_float3(fi);

	int index;
	for(int i=0;i<8;i++)
	{
		index=fi.x+(i/4) + (fi.y+(i/2))*Size.x + (fi.z+(i/1))*Size.y*Size.x;

		s[i].x=Vector_Field[index + 0];
		s[i].y=Vector_Field[index + 1];
		s[i].z=Vector_Field[index + 2];
	}

	return lerp(lerp(lerp(s[0],s[4],Vector.x),lerp(s[2],s[6],Vector.x),Vector.y),
				lerp(lerp(s[1],s[5],Vector.x),lerp(s[3],s[7],Vector.x),Vector.y),Vector.z);	
}

__global__ void IntegrateVectorField(float *Vector_Field, float3 *posptr, float *timeptr, unsigned int ElementSize, unsigned int Size_x, unsigned int Size_y, unsigned int Size_z, float stepsize, unsigned int bitmask)
{
	const int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index>ElementSize)
		return;

	int3 Size;
	Size.x=Size_x;
	Size.y=Size_y;
	Size.z=Size_z;

	float3 clVs,clVertex;

	clVertex=posptr[index];
	timeptr[index]++;//add the amount of time

	clVs=(bitmask & 0x00000001) ? Sample(clVertex,Vector_Field,Size) : SampleL(clVertex,Vector_Field,Size);

	if(bitmask & 0x00010000)
		clVertex+=(stepsize * clVs);
	else
	{
		float3 clVertexTMP=clVertex+stepsize * clVs;
		clVertex+=(0.5f*stepsize * clVs);
		clVs=(bitmask & 0x00000001) ? Sample(clVertex,Vector_Field,Size) : SampleL(clVertex,Vector_Field,Size);
		clVertex+=(0.5f*stepsize * clVs);
		clVertex=2 * clVertex-clVertexTMP;
	}

	posptr[index]=clVertex;
}

extern "C" void integrateVectorFieldGPU(float *fVectorField, float3 *posptr, float *timeptr, unsigned int uiElementSize, unsigned int uiGridSize, unsigned int uiBlockSize, unsigned int iSizeFieldx, unsigned int iSizeFieldy, unsigned int iSizeFieldz, float stepsize, unsigned int bitmask)
{
	dim3 BlockSize;
	BlockSize.x=uiBlockSize;
	dim3 GridSize;
	GridSize.x=uiGridSize;

	IntegrateVectorField<<<GridSize,BlockSize>>>(fVectorField, posptr,timeptr,uiElementSize,iSizeFieldx,iSizeFieldy,iSizeFieldz,stepsize,bitmask);
}
