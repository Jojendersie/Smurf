#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//
// Smurf
// =====
// ##### Martin Kirst, Johannes Jendersie, Christoph L�mmerhirt, Laura Osten #####
//
// Smoke Surfaces: An Interactive Flow Visualization
// Technique Inspired by Real-World Flow Experiments
//
// File:              /src/cudaintegration.cu
// Author:            Christoph L�mmerhirt
// Creation Date:     2012.01.11
// Description:
//
// Declaration of the interface from C++ to Cuda and the Cuda-Kernel.
//
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// Preprocessor Directives and Namespaces
////////////////////////////////////////////////////////////////////////////////

#include <math.h>
#include "cudamath.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

texture<float4,3,hipReadModeElementType> tex;
hipArray *d_fieldArray;

__device__ int3 convert_int3(float3 vec)
{
	int3 tmp;
	tmp.x=(int)vec.x;
	tmp.y=(int)vec.y;
	tmp.z=(int)vec.z;
	return tmp;
}

__device__ float3 convert_float3(int3 vec)
{
	float3 tmp;
	tmp.x=(float)vec.x;
	tmp.y=(float)vec.y;
	tmp.z=(float)vec.z;
	return tmp;
}

__device__ float3 Sample(float3 Vector, const float *Vector_Field, uint3 Size)
{
	int3 fi;
	int index;

	fi=convert_int3(Vector);
	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
			return make_float3(0,0,0);

	index=fi.x+fi.y*Size.x+fi.z*Size.y*Size.x;

	return make_float3(Vector_Field[(index*3)+0],Vector_Field[(index*3)+1],Vector_Field[(index*3)+2]);

	//float4 tmperg=tex3D(tex,fi.x,fi.y,fi.z);

	//return make_float3(tmperg.x,tmperg.y,tmperg.z);
}

__device__ float3 SampleL(float3 Vector, const float *Vector_Field, uint3 Size)
{
	float3 s[8];

	int3 fi;
	fi=convert_int3(Vector);

	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
			return make_float3(0,0,0);

	Vector=Vector-convert_float3(fi);

	int index;
	for(int i=0;i<8;i++)
	{
		index=fi.x+(i&1) + (fi.y+(i&2))*Size.x + (fi.z+(i&4))*Size.y*Size.x;

		s[i].x=Vector_Field[(index*3) + 0];
		s[i].y=Vector_Field[(index*3) + 1];
		s[i].z=Vector_Field[(index*3) + 2];

		//float4 tmperg=tex3D(tex,fi.x+(i&1),fi.y+(i&2),fi.z+(i&4));

		//s[i]= make_float3(tmperg.x,tmperg.y,tmperg.z);
	}

	return lerp(lerp(lerp(s[0],s[4],Vector.x),lerp(s[2],s[6],Vector.x),Vector.y),
				lerp(lerp(s[1],s[5],Vector.x),lerp(s[3],s[7],Vector.x),Vector.y),Vector.z);	
}

#define MAXRANDINV 0.00000000002328306437f
#define RNDMID 0.028f

__global__ void IntegrateVectorField(float *Vector_Field, float3 *posptr, unsigned int ElementSize, uint3 Size, uint3 rand, float3 bbMin,
									float3 posGridOffset, int resetcolumn, int rows, float stepsize, unsigned int bitmask, float avgVecLength)
{
	const int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index>ElementSize || rows*resetcolumn<index)
		return;
	
	float3 clVs,clVertex;

	clVertex=(posptr[index]-bbMin)*posGridOffset;

	clVs=(bitmask & 0x00000001) ? Sample(clVertex,Vector_Field,Size) : SampleL(clVertex,Vector_Field,Size);

	if(bitmask & 0x00001000)
	{
		float3 rnd;
		rnd.x=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);
		rnd.y=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);
		rnd.z=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);

		clVs+= avgVecLength * make_float3(((rand.z+rnd.x)*MAXRANDINV-RNDMID),((rand.z+rnd.x)*MAXRANDINV-RNDMID),((rand.z+rnd.x)*MAXRANDINV-RNDMID));
	}

	if(bitmask & 0x00010000)
		clVertex+=(stepsize * clVs);
	else
	{
		float3 clVertexTMP=clVertex+stepsize * clVs;
		clVertex+=(0.5f*stepsize * clVs);
		clVs=(bitmask & 0x00000001) ? Sample(clVertex,Vector_Field,Size) : SampleL(clVertex,Vector_Field,Size);
		clVertex+=(0.5f*stepsize * clVs);
		clVertex=2 * clVertex-clVertexTMP;
	}

	posptr[index]=clVertex/posGridOffset+bbMin;
}

__device__ float3 Sample4D(float tInterpolate, unsigned int t0, float3 Vector, const float *Vector_Field, uint4 Size)
{
	int3 fi;
	int index;

	fi=convert_int3(Vector);
	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
		return make_float3(0,0,0);

	index=fi.x+fi.y*Size.x+fi.z*Size.y*Size.x+t0*Size.x*Size.y*Size.z;

	return make_float3(Vector_Field[(index*3)+0],Vector_Field[(index*3)+1],Vector_Field[(index*3)+2]);

	//float4 tmperg=tex3D(tex,fi.x,fi.y,fi.z);

	//return make_float3(tmperg.x,tmperg.y,tmperg.z);
}

__device__ float3 SampleL4D(float tInterpolate, unsigned int t[2], float3 Vector, const float *Vector_Field, uint4 Size)
{
	float3 s[2][8];
	float3 erg[2];

	int3 fi;
	fi=convert_int3(Vector);

	if(fi.x > Size.x || fi.y > Size.y || fi.z > Size.z || fi.x<0 || fi.y<0 || fi.z<0)
		return make_float3(0,0,0);

	Vector=Vector-convert_float3(fi);

	int index;

	for(int j=0;j<2;j++)
	{
		for(int i=0;i<8;i++)
		{
			index=fi.x+(i&1) + (fi.y+(i&2))*Size.x + (fi.z+(i&4))*Size.y*Size.x+t[j]*Size.x*Size.y*Size.z;

			s[j][i].x=Vector_Field[(index*3) + 0];
			s[j][i].y=Vector_Field[(index*3) + 1];
			s[j][i].z=Vector_Field[(index*3) + 2];
			//float4 tmperg=tex3D(tex,fi.x+(i&1),fi.y+(i&2),fi.z+(i&4));

			//s[j][i]= make_float3(tmperg.x,tmperg.y,tmperg.z);
		}

		erg[j] = lerp(lerp(lerp(s[j][0],s[j][4],Vector.x),lerp(s[j][2],s[j][6],Vector.x),Vector.y),
					  lerp(lerp(s[j][1],s[j][5],Vector.x),lerp(s[j][3],s[j][7],Vector.x),Vector.y),Vector.z);	
	}

	return lerp(erg[0],erg[1],tInterpolate);
}

__global__ void IntegrateVectorField4D(float *Vector_Field, float3 *posptr, unsigned int ElementSize, uint4 Size, uint3 rand, float3 bbMin,
									   float3 posGridOffset, int resetcolumn, int rows, float stepsize, unsigned int bitmask, float avgVecLength, uint2 t, float tInterpolate)
{
	const int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index>ElementSize || rows*resetcolumn<index)
		return;
	
	float3 clVs,clVertex;

	clVertex=(posptr[index]-bbMin)*posGridOffset;

	clVs=(bitmask & 0x00000001) ? Sample4D(tInterpolate,t.x,clVertex,Vector_Field,Size) : SampleL4D(tInterpolate,(unsigned int*)&t,clVertex,Vector_Field,Size);

	if(bitmask & 0x00001000)
	{
		float3 rnd;
		rnd.x=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);
		rnd.y=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);
		rnd.z=random(index+rand.x,(clVs.x+clVs.y+clVs.z)*1000+rand.y);

		clVs+= avgVecLength * make_float3(((rand.z+rnd.x)*MAXRANDINV-RNDMID),((rand.z+rnd.x)*MAXRANDINV-RNDMID),((rand.z+rnd.x)*MAXRANDINV-RNDMID));
	}

	if(bitmask & 0x00010000)
		clVertex+=(stepsize * clVs);
	else
	{
		float3 clVertexTMP=clVertex+stepsize * clVs;
		clVertex+=(0.5f*stepsize * clVs);
		clVs=(bitmask & 0x00000001) ? Sample4D(tInterpolate,t.x,clVertex,Vector_Field,Size) : SampleL4D(tInterpolate,(unsigned int*)&t,clVertex,Vector_Field,Size);
		clVertex+=(0.5f*stepsize * clVs);
		clVertex=2 * clVertex-clVertexTMP;
	}

	posptr[index]=clVertex/posGridOffset+bbMin;
}

extern "C" void integrateVectorFieldGPU(float* fVectorField, float3 *posptr, unsigned int uiElementSize, unsigned int uiGridSize, 
										unsigned int uiBlockSize, uint4 sizeField, uint3 rnd, float3 bbMin, float3 posGridOff, 
										int resetcolumn, int rows, float stepsize, unsigned int bitmask, float avgVecSize, float tInterpolate, uint2 t)
{
	if(bitmask & 0x00000100)
		IntegrateVectorField4D<<<uiGridSize,uiBlockSize>>>(fVectorField, posptr,uiElementSize,sizeField,rnd,bbMin,posGridOff,resetcolumn,rows,stepsize,bitmask, avgVecSize,t,tInterpolate);
	else
		IntegrateVectorField<<<uiGridSize,uiBlockSize>>>(fVectorField, posptr,uiElementSize,make_uint3(sizeField.x,sizeField.y,sizeField.z),rnd,bbMin,posGridOff,resetcolumn,rows,stepsize,bitmask, avgVecSize);
}

__global__ void ResetColumn(float3* posptr, float3 bbMin, float3 bbMax, int rows, int resetColumn)
{
	const int index=threadIdx.x;
	posptr[resetColumn*rows+index]=lerp(bbMin,bbMax,index/float(rows-1));
}

extern "C" void resetOldColumn(float3* posptr, float3 bbMin, float3 bbMax, int rows, int resetColumn)
{
	ResetColumn<<<1,rows>>>(posptr,bbMin,bbMax,rows,resetColumn);
}

extern "C" void InitCuda(float *vectorField, hipExtent size)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	hipMalloc3DArray(&d_fieldArray,&channelDesc,size);

	hipMemcpy3DParms param = {0};
	param.srcPtr=make_hipPitchedPtr((void*)vectorField,size.width*sizeof(float)*3,size.width,size.height);
	param.dstArray=d_fieldArray;
	param.extent=size;
	param.kind=hipMemcpyHostToDevice;
	hipMemcpy3D(&param);

	tex.normalized=false;
	tex.addressMode[0]=hipAddressModeWrap;
	tex.addressMode[1]=hipAddressModeWrap;
	tex.addressMode[2]=hipAddressModeWrap;

	hipBindTextureToArray(&tex,d_fieldArray,&channelDesc);
}